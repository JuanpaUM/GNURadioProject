// main.cu

#include <hip/hip_runtime.h>
#include <iostream>

extern "C" void scramble_kernel(unsigned char* pkt, int codeword_start, int size_nr, unsigned char* poly, int pl_par_sizes);

int main() {
    const int N = 256;
    unsigned char pkt[N];
    unsigned char poly[N];

    // Inicializa pkt y poly

    scramble_kernel(pkt, 0, N, poly, N);

    // Realiza otras operaciones si es necesario

    std::cout << "Kernel ejecutado con éxito" << std::endl;

    return 0;
}