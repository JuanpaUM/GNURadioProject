
#include <hip/hip_runtime.h>
// scramble_kernel.cu
__global__ void scramble_kernel(unsigned char* pkt, int codeword_start, int size_nr, unsigned char* poly, int pl_par_sizes) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < pl_par_sizes) {
        pkt[codeword_start + idx] ^= poly[idx];
    }
}
